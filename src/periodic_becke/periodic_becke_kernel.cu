#include "hip/hip_runtime.h"
#include "periodic_becke_kernel.h"

#include "../periodic_kernel_data_cu.h"

#include "../gpubox.h"

namespace PeriodicBox
{
    static __device__ double get_r(double x1, double y1, double z1, double x2, double y2, double z2)
    {
        return sqrt((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1));
    }

    static __device__ double get_one_over_r(double x1, double y1, double z1, double x2, double y2, double z2)
    {
        return rsqrt((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1));
    }

    // This is the form proposed by Becke in the original Becke weight equation
    // J. Chem. Phys. 88, 2547-2553 (1988) https://doi.org/10.1063/1.454033
    static __device__ double switch_function(double m)
    {
        m = 1.5 * m - 0.5 * m*m*m;
        m = 1.5 * m - 0.5 * m*m*m;
        m = 1.5 * m - 0.5 * m*m*m;
        return 0.5 * (1.0 - m);
    }

    static __global__ void weights_kernel(const int n_point, const double* d_point_x, const double* d_point_y, const double* d_point_z,
                                          const float4* d_atoms, const int n_atom,
                                          const int* d_point_atom_center_index, double* d_weight, const double* d_interatomic_quantities,
                                          const double switch_function_threshold, const double image_cutoff_radius,
                                          const PeriodicKernelDataReal<double> periodic_data)
    {
        const int i_point = blockDim.x * blockIdx.x + threadIdx.x;
        if (i_point >= n_point) return;

        const double point[3] { d_point_x[i_point], d_point_y[i_point], d_point_z[i_point]};
        const int i_center_atom = d_point_atom_center_index[i_point];
        const double reference_image[3] { d_atoms[i_center_atom].x, d_atoms[i_center_atom].y, d_atoms[i_center_atom].z };

        double p_sum_b = 0.0;
        double p_a = 0.0;
        for (int i = 0; i < n_atom; i++) {
            const int i_atom_1 = (i_center_atom + i) % n_atom;
            double atom_1[3] = { d_atoms[i_atom_1].x, d_atoms[i_atom_1].y, d_atoms[i_atom_1].z };
            periodic_data.move_to_same_image(reference_image, atom_1);

            const double atom1_to_reference[3] { atom_1[0] - reference_image[0], atom_1[1] - reference_image[1], atom_1[2] - reference_image[2], };
            int image1_positive_bound[3] { 0, 0, 0 };
            int image1_negative_bound[3] { 0, 0, 0 };
            periodic_data.get_cube_bound_real(image1_positive_bound, image1_negative_bound, atom1_to_reference, image_cutoff_radius);

            for (int i_image1_x = -image1_negative_bound[0]; i_image1_x <= image1_positive_bound[0]; i_image1_x++)
                for (int i_image1_y = -image1_negative_bound[1]; i_image1_y <= image1_positive_bound[1]; i_image1_y++)
                    for (int i_image1_z = -image1_negative_bound[2]; i_image1_z <= image1_positive_bound[2]; i_image1_z++) {
                        double lattice_image_1[3];
                        periodic_data.get_absolute_coord_real(lattice_image_1, i_image1_x, i_image1_y, i_image1_z);
                        const double atom_image_1[3] = { atom_1[0] + lattice_image_1[0], atom_1[1] + lattice_image_1[1], atom_1[2] + lattice_image_1[2] };

                        const double ra = get_r(atom_image_1[0], atom_image_1[1], atom_image_1[2], point[0], point[1], point[2]);

                        double p_b = 1.0;
                        for (int i_atom_2 = 0; i_atom_2 < n_atom; i_atom_2++) {
                            const double a_ab = d_interatomic_quantities[i_atom_1 * n_atom + i_atom_2];
                            double atom_2[3] = { d_atoms[i_atom_2].x, d_atoms[i_atom_2].y, d_atoms[i_atom_2].z };
                            periodic_data.move_to_same_image(reference_image, atom_2);

                            const double atom2_to_reference[3] { atom_2[0] - reference_image[0], atom_2[1] - reference_image[1], atom_2[2] - reference_image[2], };
                            int image2_positive_bound[3] { 0, 0, 0 };
                            int image2_negative_bound[3] { 0, 0, 0 };
                            periodic_data.get_cube_bound_real(image2_positive_bound, image2_negative_bound, atom2_to_reference, image_cutoff_radius);

                            for (int i_image2_x = -image2_negative_bound[0]; i_image2_x <= image2_positive_bound[0]; i_image2_x++)
                                for (int i_image2_y = -image2_negative_bound[1]; i_image2_y <= image2_positive_bound[1]; i_image2_y++)
                                    for (int i_image2_z = -image2_negative_bound[2]; i_image2_z <= image2_positive_bound[2]; i_image2_z++) {
                                        double lattice_image_2[3];
                                        periodic_data.get_absolute_coord_real(lattice_image_2, i_image2_x, i_image2_y, i_image2_z);
                                        const double atom_image_2[3] = { atom_2[0] + lattice_image_2[0], atom_2[1] + lattice_image_2[1], atom_2[2] + lattice_image_2[2] };

                                        if ( (i_atom_1 != i_atom_2) ||
                                            !(i_image1_x == i_image2_x && i_image1_y == i_image2_y && i_image1_z == i_image2_z) ) {
                                            const double one_over_rab = get_one_over_r(atom_image_1[0], atom_image_1[1], atom_image_1[2], atom_image_2[0], atom_image_2[1], atom_image_2[2]);
                                            const double rb = get_r(atom_image_2[0], atom_image_2[1], atom_image_2[2], point[0], point[1], point[2]);
                                            const double mu = (ra - rb) * one_over_rab;
                                            // Refer to equation A2 in the original Becke paper for the next equation
                                            const double nu = mu + a_ab * (1.0 - mu * mu);
                                            p_b *= switch_function(nu);
                                            if (p_b < switch_function_threshold) {
                                                p_b = 0.0;
                                                goto jump_out_atom2_image_loop;
                                            }
                                        }
                                    }
                        }

                        jump_out_atom2_image_loop:
                        p_sum_b += p_b;
                        if (i_atom_1 == i_center_atom && i_image1_x == 0 && i_image1_y == 0 && i_image1_z == 0) {
                            p_a = p_b;
                            if (p_a == 0.0)
                                goto jump_out_atom1_image_loop;
                        }
                    }
        }

        jump_out_atom1_image_loop:
        double wt;
        if (p_a == 0.0)
            wt = 0.0;
        else
            wt = d_weight[i_point] * (p_a / p_sum_b);

        d_weight[i_point] = wt;
    }

    void weights(const int n_grid, const int n_block,
                 const int n_point, const double* d_point_x, const double* d_point_y, const double* d_point_z,
                 const float4* d_atoms, const int n_atom, const int* d_point_atom_center_index,
                 double* d_weight, const double* d_interatomic_quantities,
                 const double switch_function_threshold, const double image_cutoff_radius,
                 const LatticeVector unit_cell)
    {
        const PeriodicKernelDataReal<double> periodic_data(NAN, NAN, NAN, unit_cell);
        weights_kernel<<<n_grid, n_block>>>(n_point, d_point_x, d_point_y, d_point_z,
                                            d_atoms, n_atom, d_point_atom_center_index,
                                            d_weight, d_interatomic_quantities,
                                            switch_function_threshold, image_cutoff_radius, periodic_data);
    }

    static __global__ void interatomic_quantity_kernel(const float4* atoms, const int n_atom, double* d_interatomic_quantities)
    {
        const int i_atom_1 = threadIdx.x + blockIdx.x * blockDim.x;
        const int i_atom_2 = threadIdx.y + blockIdx.y * blockDim.y;

        if (i_atom_1 >= n_atom || i_atom_2 >= n_atom) return;

        const float4 atom_1 = atoms[i_atom_1];
        const float4 atom_2 = atoms[i_atom_2];

        double a_ab;
        // Refer to equation A3~A6 in the original Becke paper for the following equations
        const double chi = (double)atom_1.w / (double)atom_2.w;
        const double uab = (chi - 1.0) / (chi + 1.0);
        a_ab = uab / (uab * uab - 1.0);
        if (a_ab >  0.5)  a_ab =  0.5;
        if (a_ab < -0.5)  a_ab = -0.5;

        d_interatomic_quantities[i_atom_1 * n_atom + i_atom_2] = a_ab;
    }

    void set_atom_and_interatomic_quantities(GPUBox* gpu, double*& d_interatomic_quantities, float4*& d_atoms,
                                             const int n_atom, const double* atom_xyz, const double* atom_radius)
    {
        float4* h_atoms = (float4*)gpu->cpuAlloc(sizeof(float4) * n_atom);
        d_atoms = (float4*)gpu->gpuAlloc(sizeof(float4) * n_atom);

        for (int i = 0; i < n_atom; i++) {
            h_atoms[i].x = (float)atom_xyz[3 * i + 0];
            h_atoms[i].y = (float)atom_xyz[3 * i + 1];
            h_atoms[i].z = (float)atom_xyz[3 * i + 2];
            h_atoms[i].w = (float)atom_radius[i];
        }

        hipMemcpy(d_atoms, h_atoms, n_atom * sizeof(float4), hipMemcpyHostToDevice);
        gpu->cpuFree(h_atoms);

        const dim3 block_dimension(16, 16);
        const dim3 grid_dimension((n_atom + block_dimension.x - 1) / block_dimension.x, (n_atom + block_dimension.y - 1) / block_dimension.y, 1);

        d_interatomic_quantities = (double*)gpu->gpuAlloc(sizeof(double) * n_atom * n_atom);
        interatomic_quantity_kernel<<<grid_dimension, block_dimension>>>(d_atoms, n_atom, d_interatomic_quantities);
    }

    // For gradient

//     __global__ void Wgrad_cache(const int numPoints, const int numAtoms, const int grad_pitch,
//                                 const double* xpts, const double* ypts, const double* zpts, const float4* d_atoms,
//                                 double* ra_cache, double* p_cache,
//                                 const double2* cache, const double switch_function_threshold)
//     {
//         int ptIndex = threadIdx.x+blockIdx.x*WGRAD_BLOCK_XDIM_DP;
//         if( ptIndex >= numPoints )    return;
//         int a = threadIdx.y + blockIdx.y*WGRAD_BLOCK_YDIM_DP;
//         if( a >= numAtoms )    return;
//         double ptx = xpts[ptIndex];
//         double pty = ypts[ptIndex];
//         double ptz = zpts[ptIndex];
//         float4 temp = d_atoms[a];

//         double ra = get_r(temp.x, temp.y, temp.z, ptx, pty, ptz);
//         ra_cache[ptIndex+grad_pitch*a] = ra;

//         double pVal = 1.0;
//         double2 buff = cache[numAtoms*a];

//         for(int b=0; b<numAtoms; ++b)
//         {
//             double2 reg = buff;
//             int next = b+1;
//             buff = cache[a*numAtoms+next];

//             if( a == b )    continue;

//             temp = d_atoms[b];
//             double rb = get_r(temp.x, temp.y, temp.z, ptx, pty, ptz);
//             double mu = (ra - rb) * reg.x;
//             double nu = mu + reg.y*(1.0 - mu*mu);
//             pVal *= switch_function(nu);
//             if( fabs(pVal) < switch_function_threshold ){
//                 pVal = 0.0;
//                 break;
//             }
//         }
//         p_cache[ptIndex+grad_pitch*a] = pVal;
//     }

//     void Wgrad_cache_caller(const dim3 GRID, const dim3 BLOCK, const int numPoints, const int numAtoms, const int grad_pitch,
//                             const double* xpts, const double* ypts, const double* zpts, const float4* d_atoms,
//                             double* ra_cache, double* p_cache,
//                             const double2* cache, const double switch_function_threshold)
//     {
//         Wgrad_cache<<<GRID, BLOCK>>>(numPoints, numAtoms, grad_pitch, xpts, ypts, zpts, d_atoms, ra_cache, p_cache, cache, switch_function_threshold);
//     }

//     __device__ double3 Dev_grad_nu(int g, double3 rg_vec, double rg, int b, double rb, double d_gb, double a_gb, const float4* d_atoms)
//     {
//         double3 grad;

//         double3 r_gb;
//         r_gb.x = d_atoms[g].x - d_atoms[b].x;
//         r_gb.y = d_atoms[g].y - d_atoms[b].y;
//         r_gb.z = d_atoms[g].z - d_atoms[b].z;

//         double mu = (rg - rb)*d_gb;
//         double coef = 1.0-2.0*a_gb*mu;
//         double r_g_coef;

//         if (rg < 1.0e-14)     r_g_coef = 0.0;
//         else                  r_g_coef = -coef*d_gb/rg;
//         grad.x = r_g_coef * rg_vec.x;
//         grad.y = r_g_coef * rg_vec.y;
//         grad.z = r_g_coef * rg_vec.z;

//         double r_gb_coef = -coef*mu*d_gb*d_gb;
//         grad.x += r_gb_coef * r_gb.x;
//         grad.y += r_gb_coef * r_gb.y;
//         grad.z += r_gb_coef * r_gb.z;
//         return grad;
//     }

    static __device__ double switch_function_derivative(const double m)
    {
        const double f1 = 1.5 *  m - 0.5 * m*m*m;
        const double f2 = 1.5 * f1 - 0.5 * f1*f1*f1;
        const double f3 = 1.5 * f2 - 0.5 * f2*f2*f2;
        const double s = 0.5 * (1.0 - f3);
        if (fabs(s) < 1.0e-14)
            return 0.0;
        else
            return -(27.0/16.0) * (1.0 - f2*f2) * (1.0 - f1*f1) * (1.0 - m*m) / s;
    }
//     __device__ double Dev_comp_t(double mu_ij, double a_ij)
//     {
//         double t;
//         double nu = mu_ij + a_ij*(1.0-mu_ij*mu_ij);
//         double p1 = 1.5*nu - 0.5*nu*nu*nu;
//         double p2 = 1.5*p1 - 0.5*p1*p1*p1;
//         double s_val;
//         double p3 = 1.5*p2 - 0.5*p2*p2*p2;
//         s_val = 0.5 * (1.0 - p3);
//         if (fabs(s_val) < 1.0e-14)
//             return 0.0;
//         t = -(27.0/16.0) * (1.0 - p2*p2) * (1.0 - p1*p1) * (1.0 - nu*nu) / s_val;
//         return t;
//     }

//     __global__ void Wgrad_kernel(const int numPoints, const int numAtoms, const int grad_pitch,
//                                 const double* xpts, const double* ypts, const double* zpts, const double* wpts,
//                                 const float4* d_atoms, const int* ctrs,
//                                 double* xGrad, double* yGrad, double* zGrad,
//                                 const double* ra_cache, const double* p_cache, const double2* cache)
//     {
//         int ptIndex = threadIdx.x+blockIdx.x*WGRAD_BLOCK_XDIM_DP;
//         if( ptIndex >= numPoints )    return;
//         int ctr = ctrs[ptIndex];
//         int grad_atom = threadIdx.y + blockIdx.y*WGRAD_BLOCK_YDIM_DP;
//         if( grad_atom >= ctr )     ++grad_atom;
//         if( grad_atom >= numAtoms )    return;
//         double ptx = xpts[ptIndex];
//         double pty = ypts[ptIndex];
//         double ptz = zpts[ptIndex];
//         double ptw = wpts[ptIndex];
//         float4 tAtom;

//         double p_buff = p_cache[ptIndex];
//         double p_sum = 0.0;
//         double p_a = p_cache[ptIndex + ctr       * grad_pitch];
//         double p_g = p_cache[ptIndex + grad_atom * grad_pitch];
//         double3 grad_c_w_a;    grad_c_w_a.x = grad_c_w_a.y = grad_c_w_a.z = 0.0f;
//         double3 grad_c_p_a;    grad_c_p_a.x = grad_c_p_a.y = grad_c_p_a.z = 0.0f;
//         double3 grad_c_p_g;    grad_c_p_g.x = grad_c_p_g.y = grad_c_p_g.z = 0.0f;
//         tAtom = d_atoms[grad_atom];
//         double3 rg_vec;
//         rg_vec.x = ptx - tAtom.x;
//         rg_vec.y = pty - tAtom.y;
//         rg_vec.z = ptz - tAtom.z;
//         double rg = sqrt(rg_vec.x*rg_vec.x + rg_vec.y*rg_vec.y + rg_vec.z*rg_vec.z);

//         int next = ptIndex;
//         for (int bcenter=0; bcenter<numAtoms; ++bcenter)
//         {
//             double3 grad_c_p_b;
//             double t;

//             next += grad_pitch;
//             double p_b = p_buff;     p_buff = p_cache[next];
//             p_sum += p_b;
//             if( (p_b == 0.0f && p_g == 0.0f) || grad_atom == bcenter )
//                 continue;

//             double rb  = ra_cache[ptIndex+bcenter*grad_pitch];
//             double2 bg = cache[bcenter*numAtoms+grad_atom];
//             double mu_gb = (rg - rb) * bg.x;
//             double3 grad_nu = Dev_grad_nu(grad_atom, rg_vec, rg, bcenter, rb, bg.x, -bg.y, d_atoms);

//             if( p_g != 0.0f )
//             {
//                 t = Dev_comp_t(mu_gb, -bg.y);
//                 grad_c_p_g.x += t*grad_nu.x;
//                 grad_c_p_g.y += t*grad_nu.y;
//                 grad_c_p_g.z += t*grad_nu.z;
//             }

//             if(p_b != 0.0f)
//             {
//                 t = -Dev_comp_t(-mu_gb, bg.y)*p_b;
//                 grad_c_p_b.x = t * grad_nu.x;
//                 grad_c_p_b.y = t * grad_nu.y;
//                 grad_c_p_b.z = t * grad_nu.z;

//                 if(bcenter == ctr)
//                     grad_c_p_a = grad_c_p_b;

//                 grad_c_w_a.x -= grad_c_p_b.x;
//                 grad_c_w_a.y -= grad_c_p_b.y;
//                 grad_c_w_a.z -= grad_c_p_b.z;
//             }
//         }
//         grad_c_w_a.x -= grad_c_p_g.x*p_g;
//         grad_c_w_a.y -= grad_c_p_g.y*p_g;
//         grad_c_w_a.z -= grad_c_p_g.z*p_g;

//         double coef1 = p_a / p_sum;
//         double coef2 = ptw / p_sum;
//         xGrad[ptIndex+grad_atom*grad_pitch] += coef2*(coef1*grad_c_w_a.x + grad_c_p_a.x);
//         yGrad[ptIndex+grad_atom*grad_pitch] += coef2*(coef1*grad_c_w_a.y + grad_c_p_a.y);
//         zGrad[ptIndex+grad_atom*grad_pitch] += coef2*(coef1*grad_c_w_a.z + grad_c_p_a.z);
//     }

//     void Wgrad_kernel_caller(const dim3 GRID, const dim3 BLOCK, const int numPoints, const int numAtoms, const int grad_pitch, 
//                             const double* xpts, const double* ypts, const double* zpts, const double* wpts,
//                             const float4* d_atoms, const int* ctrs,
//                             double* xGrad, double* yGrad, double* zGrad,
//                             const double* ra_cache, const double* p_cache, const double2* cache)
//     {
//         Wgrad_kernel<<<GRID, BLOCK>>>(numPoints, numAtoms, grad_pitch, xpts, ypts, zpts, wpts, d_atoms,
//                                     ctrs, xGrad, yGrad, zGrad, ra_cache, p_cache, cache);
//     }

//     __global__ void Wgrad_ptctr(const int numPoints, const int numAtoms, const int grad_pitch, const int* ctrs,
//                                 double* xGrad, double* yGrad, double* zGrad)
//     {
//         double x=0.0, y=0.0, z=0.0;
//         int ptIndex = threadIdx.x + blockIdx.x*WGRAD_PTCTR_BLOCK;
//         if(ptIndex >= numPoints)    return;
//         int ctr = ctrs[ptIndex];

//         for(int j=0; j<numAtoms; ++j){
//             x -= xGrad[ptIndex+j*grad_pitch];
//             y -= yGrad[ptIndex+j*grad_pitch];
//             z -= zGrad[ptIndex+j*grad_pitch];
//         }

//         xGrad[ptIndex+ctr*grad_pitch] += x;
//         yGrad[ptIndex+ctr*grad_pitch] += y;
//         zGrad[ptIndex+ctr*grad_pitch] += z;
//     }

//     void Wgrad_ptctr_caller(const int GRID, const int BLOCK, const int numPoints, const int numAtoms, const int grad_pitch, const int* ctrs,
//                             double* xGrad, double* yGrad, double* zGrad)
//     {
//         Wgrad_ptctr<<<GRID, BLOCK>>>(numPoints, numAtoms, grad_pitch, ctrs, xGrad, yGrad, zGrad);
//     }

//     __global__ void Wgrad_dosums(const int numPoints, double* final_grad,
//                                 const double* xGrad, const double* yGrad, const double* zGrad)
//     {
//         __shared__ double scr[WGRAD_DOSUMS_BLOCK];
//         int atom = blockIdx.x/3;
//         int dimSelect = blockIdx.x%3;
//         const double *end, *cur, *temp;
//         double sum = 0.0;
//         if( dimSelect == 0 )
//             temp = xGrad;
//         else if( dimSelect == 1 )
//             temp = yGrad;
//         else
//             temp = zGrad;
//         cur = temp + numPoints*atom + threadIdx.x;
//         end = temp + numPoints * (atom+1);

//         while(cur < end)
//         {
//             sum += *cur;
//             cur += WGRAD_DOSUMS_BLOCK;
//         }
//         scr[threadIdx.x] = sum;

//         for(int span=WGRAD_DOSUMS_BLOCK/2; span>1; span /= 2)
//         {
//             __syncthreads();
//             if(threadIdx.x < span)
//                 scr[threadIdx.x] += scr[threadIdx.x+span];
//         }

//         if( !threadIdx.x )
//         {
//             sum = scr[0]+scr[1];
//             final_grad[blockIdx.x] = sum;
//         }
//     }

//     void Wgrad_dosums_caller(const int GRID, const int BLOCK, const int numPoints, double* final_grad,
//                             const double* xGrad, const double* yGrad, const double* zGrad)
//     {
//         Wgrad_dosums<<<GRID, BLOCK>>>(numPoints, final_grad, xGrad, yGrad, zGrad);
//     }

}
